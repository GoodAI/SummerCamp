#include <hip/hip_runtime.h>

#include <math.h>

extern "C"{

	// Kernel for calculation of external output of neurons of LSM

	__global__ void LSMOutputKernel(
		float* states, // inner states of neurons
		float* nodeOutput, // output of LSM
		int* outputs, // indexes of output neurons
		float threshold, // threshold for sending of output
		int count // number of neurons
		)
	{

		int id = blockDim.x*blockIdx.y*gridDim.x
			+ blockDim.x*blockIdx.x
			+ threadIdx.x;

		if (id < count){
			int index = outputs[id];

			int c1 = (states[index] >= threshold);

			nodeOutput[id] = c1;
		}

	}

}