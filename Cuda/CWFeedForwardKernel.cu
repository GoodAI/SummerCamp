#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>

#include "C:\Users\Alka\Disk Google\fel\SummerCamp\BrainSimulator-master\Sources\Modules\BasicNodes\Cuda\NeuralNetwork\Activation/ActivationFunction.cu"


extern "C"
{
	__constant__ int D_INPUT_UNITS;
	__constant__ int D_HIDDEN_UNITS;
	__constant__ int D_OUTPUT_UNITS;
	__constant__ ActivationFunctionEnum D_ACTIVATION_FUNCTION;


	__global__ void CWFeedforwardHiddenKernel(
		float *input, 
		float *hiddenActivations,
		float *previousHiddenActivations, 
		float *hiddenActivationDerivatives, 
		float *inputWeights, 
		float *recurrentWeights,
		int* periods,
		int simulationStep)
	{
		int unitId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
			+ blockDim.x*blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (unitId < D_HIDDEN_UNITS && simulationStep % periods[unitId] == 0)
		{
			//int weightId = unitId * (1 + D_INPUT_UNITS);
			int weightId = unitId * D_INPUT_UNITS;

			float weightedSum = 0;
			for (int i = 0; i < D_INPUT_UNITS; i++)
			{
				weightedSum += inputWeights[weightId] * input[i];
				weightId++;
			}

			weightId = unitId * D_HIDDEN_UNITS;

			for (int i = 0; i < D_HIDDEN_UNITS; i++)
			{
				weightedSum += recurrentWeights[weightId] * previousHiddenActivations[i];
				weightId++;
			}

			hiddenActivations[unitId] = Evaluate(D_ACTIVATION_FUNCTION, weightedSum);
			hiddenActivationDerivatives[unitId] = EvaluateDerivative(D_ACTIVATION_FUNCTION, weightedSum);
		}
	}

	__global__ void CWFeedforwardOutputKernel(float *hiddenActivations, float *outputActivations, float *outputActivationDerivatives, float *outputWeights, int simulationStep)
	{
		int unitId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
			+ blockDim.x*blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (unitId < D_OUTPUT_UNITS)
		{
			//int weightId = unitId * (1 + D_HIDDEN_UNITS);
			int weightId = unitId * D_HIDDEN_UNITS;

			float weightedSum = 0;
			for (int i = 0; i < D_HIDDEN_UNITS; i++)
			{
				weightedSum += outputWeights[weightId] * hiddenActivations[i];
				weightId++;
			}

			outputActivations[unitId] = Evaluate(D_ACTIVATION_FUNCTION, weightedSum);
			//outputActivations[unitId] = simulationStep;
			outputActivationDerivatives[unitId] = EvaluateDerivative(D_ACTIVATION_FUNCTION, weightedSum);
		}
	}
}
