#include <hip/hip_runtime.h>

#include <math.h>

extern "C"{

	// Kernel for computing the inner state of neurons
	// The main equation for inner state of neurons X in time T is:
	// innerState[X, T] = (innerState[X, T-1] + A * imageInput + B * 1/N * sum(all edge inputs for X) / (A + B + Threshold),
	// where N is number of input neurons for neuron X and A/B are constants changeable in BrainSimulator

	__global__ void IF2ComputeStateKernel(
		int initState, // value of init state
		int refractoryState, // value of refractory state
		float refractory, // refractory
		float* edgeInputs, // edge inputs
		float* imageInput, // image inputs
		float* neuronOutputs, // output of neurons
		float* innerStates, // inner states of neurons
		float threshold, // threshold for sending of output
		float connectivity, // connectivity of the network
		int count // number of neurons
		)
	{

		int id = blockDim.x*blockIdx.y*gridDim.x
			+ blockDim.x*blockIdx.x
			+ threadIdx.x;

		if (id < count){

			neuronOutputs[id] = 0;

			if (innerStates[id] >= threshold){
				neuronOutputs[id] = innerStates[id];
				innerStates[id] = -65;
			}
			else {
				float totalInput = 0;

				for (int i = 0; i < count; i++) {
					totalInput += edgeInputs[i * count + id];
				}

				totalInput += imageInput[id];
				if (totalInput > 0){
					innerStates[id] += totalInput;
				}
			}

			/*int c1 = (innerStates[id] >= threshold);
			int c2 = (innerStates[id] < threshold);

			neuronOutputs[id] = c1 * innerStates[id];
			innerStates[id] = c2 * innerStates[id] + c1 * -130;

			int c3 = (innerStates[id] >= -65);
			int c4 = (innerStates[id] < -65);

			innerStates[id] = c3 * innerStates[id] + c4 * (innerStates[id] / 1.3f);

			if (c2 && c3){
			float totalInput = 0;

			for (int i = 0; i < count; i++) {
			totalInput += edgeInputs[i * count + id];
			}

			totalInput += imageInput[id];

			int c5 = (totalInput > 0);
			innerStates[id] += c5 * totalInput;
			}*/

			imageInput[id] = 0;
		}

	}

}