#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>



extern "C"
{
	__constant__ int D_INPUT_UNITS;
	__constant__ int D_HIDDEN_UNITS;
	__constant__ int D_OUTPUT_UNITS;



	//kernel code
	__global__ void CWOutputDeltaKernel(float *outputDeltas, float *target, float *outputActivations, float *outputActivationDerivatives)
	{
		int unitId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
			+ blockDim.x*blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;
		if (unitId < D_OUTPUT_UNITS)
		{
			outputDeltas[unitId] = (target[unitId] - outputActivations[unitId]) * outputActivationDerivatives[unitId];
		}
	}
}