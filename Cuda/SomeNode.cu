
#include "hip/hip_runtime.h"



extern "C"
{
	__global__ void IncrementAll(float* input, float* output, float incrementSize, int itemCount)
	{
		int threadId = blockIdx.y*blockDim.x*gridDim.x
			+ blockIdx.x*blockDim.x
			+ threadIdx.x;

		if (threadId < itemCount)
		{
			output[threadId] = input[threadId] + incrementSize;
		}
	}
}