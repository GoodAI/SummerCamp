#include <hip/hip_runtime.h>

#include <math.h>

extern "C"{

	// Kernel for computing the inner state of neurons
	// The main equation for neurons X in time T is:
	// innerState[X, T] = (innerState[X, T-1] + A * imageInput + B * 1/N * sum(all edge inputs for X) / (A + B + Threshold),
	// where N is number of input neurons for neuron X and A/B are constants changeable in BrainSimulator

	__global__ void LSMComputeStateKernel(
		float a, // A constant of the main equation
		float b, // B constant of the main equation
		float* edgeInputs, // edge inputs
		float* imageInput, // image inputs
		float* neuronOutputs, // output of neurons
		float* innerStates, // inner states of neurons
		float threshold, // threshold for sending of output
		float connectivity, // connectivity of the network
		int count // number of neurons
		)
	{

		int id = blockDim.x*blockIdx.y*gridDim.x
			+ blockDim.x*blockIdx.x
			+ threadIdx.x;

		if (id < count){
			float totalInput = 0;

			for (int i = 0; i < count; i++) {
				totalInput += edgeInputs[i * count + id];
			}

			totalInput = (totalInput * b) / (connectivity * count);

			totalInput += imageInput[id] * a;
			imageInput[id] = 0;

			neuronOutputs[id] = 0;

			innerStates[id] += totalInput;

			innerStates[id] /= (a + b + threshold);

			int c1 = (innerStates[id] >= threshold);
			int c2 = (innerStates[id] < threshold);

			neuronOutputs[id] = innerStates[id] * c1;
			innerStates[id] = innerStates[id] * c2;
		}

	}

}