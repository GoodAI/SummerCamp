#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>

#include "../NeuralNetwork/Activation/ActivationFunction.cu"


extern "C"
{
	__constant__ int D_INPUT_UNITS;
	__constant__ int D_HIDDEN_UNITS;
	__constant__ int D_OUTPUT_UNITS;
	__constant__ int D_NEURONS_PER_GROUP;
	__constant__ int D_NEURON_GROUPS;
	__constant__ ActivationFunctionEnum D_ACTIVATION_FUNCTION;

	__global__ void CWFeedforwardHiddenKernel(
		float *input, 
		float *hiddenActivations,
		float *hiddenActivationDerivatives, 
		float *inputWeights, 
		float *recurrentWeights,
		int *activeGroups
		)
	{
		int unitId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
			+ blockDim.x*blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		extern __shared__ float activeGroupsShared[];

		if (unitId < D_NEURON_GROUPS)
		{
			activeGroupsShared[unitId] = activeGroups[unitId];
		}
		__syncthreads();

		int groupID = unitId / D_NEURONS_PER_GROUP;

		if (unitId < D_HIDDEN_UNITS && activeGroupsShared[groupID] == 1)
		{
			int weightId = unitId * D_INPUT_UNITS;

			float weightedSum = 0;
			for (int i = 0; i < D_INPUT_UNITS; i++)
			{
				weightedSum += inputWeights[weightId] * input[i];
				weightId++;
			}

			weightId = unitId * D_HIDDEN_UNITS;

			for (int i = 0; i < D_HIDDEN_UNITS; i++)
			{
				weightedSum += recurrentWeights[weightId] * hiddenActivations[i];
				weightId++;
			}
			
			__syncthreads();

			hiddenActivations[unitId] = Evaluate(D_ACTIVATION_FUNCTION, weightedSum);
			hiddenActivationDerivatives[unitId] = EvaluateDerivative(D_ACTIVATION_FUNCTION, weightedSum);
		}
	}

	__global__ void CWFeedforwardOutputKernel(float *hiddenActivations, float *outputActivations, float *outputActivationDerivatives, float *outputWeights)
	{
		int unitId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
			+ blockDim.x*blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		if (unitId < D_OUTPUT_UNITS)
		{
			
			int weightId = unitId * D_HIDDEN_UNITS;

			float weightedSum = 0;
			for (int i = 0; i < D_HIDDEN_UNITS; i++)
			{
				weightedSum += outputWeights[weightId] * hiddenActivations[i];
				weightId++;
			}

			outputActivations[unitId] = Evaluate(D_ACTIVATION_FUNCTION, weightedSum);
			
			outputActivationDerivatives[unitId] = EvaluateDerivative(D_ACTIVATION_FUNCTION, weightedSum);
		}
	}
}
