#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>



extern "C"
{
	__constant__ int D_INPUT_UNITS;
	__constant__ int D_HIDDEN_UNITS;
	__constant__ int D_OUTPUT_UNITS;
	__constant__ int D_NEURONS_PER_GROUP;
	__constant__ int D_NEURON_GROUPS;

	__global__ void CWChangeInputWeightsKernel(
		float *inputWeights,
		float *inputWeightDeltas,
		float *outputWeights,
		float *outputDeltas,
		float *inputWeightRTRLDerivatives,

		float trainingRate,
		float momentum,

		int *activeGroups,
		int contextByActivations
		)
	{
		int weightId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
			+ blockDim.x*blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		int unitID = weightId / D_INPUT_UNITS;
		int groupID = unitID / D_NEURONS_PER_GROUP;

		extern __shared__ float activeGroupsShared[];

		if (weightId < D_NEURON_GROUPS)
		{
			activeGroupsShared[weightId] = activeGroups[weightId];
		}
		__syncthreads();

		if (weightId < D_HIDDEN_UNITS * D_INPUT_UNITS 
			&& (contextByActivations || activeGroupsShared[groupID]))
		{
			float gradient = 0;
			for (int i = 0; i < D_OUTPUT_UNITS; i++)
			{
				float sum = 0;
				for (int j = 0; j < D_HIDDEN_UNITS; j++)
				{
					sum += outputWeights[i * D_HIDDEN_UNITS + j] * inputWeightRTRLDerivatives[j * D_HIDDEN_UNITS * D_INPUT_UNITS + weightId];
				}

				gradient += outputDeltas[i] * sum;
			}

			__syncthreads();

			float weightDelta = trainingRate * gradient + momentum * inputWeightDeltas[weightId];
			inputWeightDeltas[weightId] = weightDelta;
			inputWeights[weightId] += weightDelta;
		}
	}

	__global__ void CWChangeRecurrentWeightsKernel(
		float *recurrentWeights,
		float *recurrentWeightDeltas,
		float *outputWeights,
		float *outputDeltas,
		float *recurrentWeightRTRLDerivatives,

		float trainingRate,
		float momentum,

		int *activeGroups,
		int contextByActivations
		)
	{
		int weightId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
			+ blockDim.x*blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		int unitID = weightId / D_HIDDEN_UNITS;
		int groupID = unitID / D_NEURONS_PER_GROUP;
		int x = weightId - (unitID * D_HIDDEN_UNITS);

		extern __shared__ float activeGroupsShared[];

		if (weightId < D_NEURON_GROUPS)
		{
			activeGroupsShared[weightId] = activeGroups[weightId];
		}
		__syncthreads();

		if (weightId < D_HIDDEN_UNITS * D_HIDDEN_UNITS 
			&& (contextByActivations || activeGroupsShared[groupID])
			&& (contextByActivations || (x >= (groupID * D_NEURONS_PER_GROUP))))		
		{
			float gradient = 0;

			for (int i = 0; i < D_OUTPUT_UNITS; i++)
			{
				float sum = 0;
				for (int j = 0; j < D_HIDDEN_UNITS; j++)
				{
					sum += outputWeights[i * D_HIDDEN_UNITS + j] * recurrentWeightRTRLDerivatives[j * D_HIDDEN_UNITS * D_HIDDEN_UNITS + weightId];
				}

				gradient += outputDeltas[i] * sum;
			}

			float weightDelta = trainingRate * gradient + momentum * recurrentWeightDeltas[weightId];
			recurrentWeightDeltas[weightId] = weightDelta;
			recurrentWeights[weightId] += weightDelta;
		}
	}

	__global__ void CWChangeOutputWeightsKernel(
		float *outputWeights,
		float *outputWeightDeltas,
		float *outputDeltas,
		float *hiddenActivations,

		float trainingRate,
		float momentum
		)
	{
		int weightId = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
			+ blockDim.x*blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;

		int to = weightId / D_HIDDEN_UNITS;
		int from = weightId % D_HIDDEN_UNITS;

		if (weightId < D_OUTPUT_UNITS * D_HIDDEN_UNITS)
		{
			float gradient = outputDeltas[to] * hiddenActivations[from];
			float weightDelta = trainingRate * gradient + momentum * outputWeightDeltas[weightId];
			outputWeightDeltas[weightId] = weightDelta;
			outputWeights[weightId] += weightDelta;
		}
	}
}
