#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <math.h>



extern "C"
{
	__constant__ int D_NEURONS_PER_GROUP;
	__constant__ int D_NEURONS_GROUPS;
	__constant__ int D_HIDDEN_UNITS;

	//kernel code
	__global__ void CWGetContext(float *activations, float *contextActivations)
	{
		int id = blockDim.x*blockIdx.y*gridDim.x	//rows preceeding current row in grid
			+ blockDim.x*blockIdx.x				//blocks preceeding current block
			+ threadIdx.x;
		
		if (id < D_NEURONS_PER_GROUP*D_NEURONS_GROUPS)
		{
			int unitID = id % D_HIDDEN_UNITS;
			int unitPeriod = unitID / D_NEURONS_PER_GROUP + 1;
			int groupPeriod = id / D_HIDDEN_UNITS;
			contextActivations[id] += activations[unitID] * (unitPeriod / groupPeriod);
		}
	}
}